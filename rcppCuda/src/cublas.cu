/* CUDA API header files*/
#include "hip/hip_runtime.h"



// CUBLAS library
#include "hipblas.h"

extern "C"
void amax(double *x, int *index, int *size) {
	// Initialise a few things
	hipError_t cudaStat; // cudaMalloc status
	hipblasStatus_t stat; // CUBLAS functions status
	hipblasHandle_t handle; // CUBLAS context
	
	// Copy passed argument on the device
	double *xd;
	cudaStat = hipMalloc((void**) &xd, *size * sizeof(*x));	// Memory alloc for xd
	stat = hipblasCreate(&handle);	// Initialize CUBLAS context
	stat = hipblasSetVector(*size, sizeof(*x), x, 1, xd, 1);	//x -> xd

	// Run the CUBLAS function
	stat = hipblasIdamax(handle, *size, xd, 1, index);
	
	hipFree(xd); // free memory
	hipblasDestroy(handle); // destroy CUBLAS context
}

extern "C"
void amax_u(double *x, int *index, int *size) {
	hipblasHandle_t handle; // CUBLAS context

	// Copy passed argument on the device
	double *xd;
	hipMalloc((void**) &xd, *size * sizeof(*x));
	hipMemcpy(xd, x, *size * sizeof(*x), hipMemcpyHostToDevice);	// copy data to device
	
	// Run the CUBLAS function
	hipblasCreate(&handle);	// Initialize CUBLAS context
	hipblasIdamax(handle, *size, xd, 1, index);
	hipDeviceSynchronize(); // Blocks until the device has completed all preceding requested tasks

	hipFree(xd); // free memory
	hipblasDestroy(handle); // destroy CUBLAS context
}